#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <string.h>
#include <math.h> // For pow
#include <hip/hip_runtime.h>
#include <thread>
#include <vector>
#include <mutex>
#include <atomic>
#include <unistd.h>
#include <fstream>
#include <unordered_set>
#include <functional>
#include <iostream>
#include <queue>
#include <condition_variable>
#include <cstdlib> // For strtoull
#include <sys/stat.h>
#include <errno.h>
#include <algorithm> // For std::reverse
#include <cstdint>   // For fixed-size types
#include <chrono>

// Constants and Macros
#define BINARY_LENGTH 135
#define INITIAL_VALUE "4000000000000000000000000000000000" // Hex string representing 135-bit value
#define THREADS_PER_BLOCK 256
#define BLOCKS_PER_GPU 256
#define TARGET_TAIL_BITS 25 // Number of trailing zero bits required
#define BATCH_SIZE 1000000ULL
#define SEED 1234 // Base seed for random number generator
#define MAX_DISTINGUISHED_POINTS_PER_KERNEL 1000000 // Maximum DPs per kernel
#define BUFFER_SIZE 100000 // Number of Points to buffer before writing to disk
#define NUM_STEP_FUNCTIONS 64 // Increased for finer granularity
#define FILE_SIZE_CAP (500ULL * 1024ULL * 1024ULL * 1024ULL) // 500GB

// Structure Definitions

struct Steps128 {
    unsigned long long low;
    unsigned long long high;
} __attribute__((packed));

#pragma pack(push, 1)
struct Point {
    unsigned char high;        // 1 byte (bits 134–128)
    unsigned long long mid;    // 8 bytes (bits 127–64)
    unsigned long long low;    // 8 bytes (bits 63–0)
    Steps128 steps;            // 16 bytes
    unsigned char is_tame;     // 1 byte
} __attribute__((packed));
#pragma pack(pop)

// Static assertion to ensure Point struct size is consistent
static_assert(sizeof(Point) == 34, "Point struct size must be 34 bytes");

// Custom hash function for Point
struct PointHash {
    std::size_t operator()(const Point& p) const {
        // Improved hash combining using prime multipliers
        std::size_t res = 17;
        res = res * 31 + std::hash<unsigned char>()(p.high);
        res = res * 31 + std::hash<unsigned long long>()(p.mid);
        res = res * 31 + std::hash<unsigned long long>()(p.low);
        return res;
    }
};

// Custom equality function for Point
struct PointEqual {
    bool operator()(const Point& a, const Point& b) const {
        return (a.high == b.high) && (a.mid == b.mid) && (a.low == b.low);
    }
};

// Structure to define step thresholds
struct StepThreshold {
    unsigned long long high;
    unsigned long long low;
    unsigned int exponent; // For printing purposes, e.g., 60 for 2^60
    bool printed;
};

// Thread-safe queue for collision detection
template <typename T>
class ThreadSafeQueue {
private:
    std::queue<T> queue_;
    std::mutex mutex_;
    std::condition_variable cond_var_;
public:
    void enqueue(T item) {
        {
            std::lock_guard<std::mutex> lock(mutex_);
            queue_.push(std::move(item));
        }
        cond_var_.notify_one();
    }

    bool dequeue(T& item) {
        std::unique_lock<std::mutex> lock(mutex_);
        while (queue_.empty()) {
            cond_var_.wait(lock);
            if (queue_.empty()) {
                return false;
            }
        }
        if (!queue_.empty()) {
            item = std::move(queue_.front());
            queue_.pop();
            return true;
        }
        return false;
    }
};

// Structure for 128-bit counter
struct Counter128 {
    unsigned long long low;
    unsigned long long high;
} __attribute__((packed));

// Device function to atomically add to a 128-bit counter
__device__ void atomicAdd128(Counter128 *counter, Steps128 value)
{
    unsigned long long old_low = atomicAdd(&(counter->low), value.low);
    unsigned long long carry = (old_low + value.low < old_low) ? 1ULL : 0ULL;
    atomicAdd(&(counter->high), value.high + carry);
}

// Function Prototypes
void step_monitor(Counter128 *global_counter, std::vector<StepThreshold> &thresholds, std::atomic<bool> &stop_flag, std::mutex &dp_mutex, Point &last_tame_dp, Point &last_wild_dp, std::atomic<unsigned long long> &kernel_launch_count, int device_count);
void collision_detection_thread(
    ThreadSafeQueue<Point> &collision_queue,
    std::unordered_set<Point, PointHash, PointEqual> &tame_points,
    std::mutex &collision_mutex,
    std::atomic<bool> &collision_found
);
void run_on_device(
    int device_id,
    Counter128 *global_counter,
    std::atomic<unsigned long long> &global_dp_counter,
    ThreadSafeQueue<Point> &collision_queue,
    std::atomic<bool> &collision_found,
    std::mutex &dp_mutex,
    Point &last_wild_dp,
    std::atomic<unsigned long long> &kernel_launch_count
);
void compute_tame_path(
    const char* initial_value,
    std::unordered_set<Point, PointHash, PointEqual> &tame_points,
    std::atomic<bool> &collision_found,
    std::mutex &dp_mutex,
    Point &last_tame_dp
);
void write_tame_dp_to_file(const Point &p, int device_id, int &file_part_number, unsigned long long &current_file_size, std::vector<Point> &tame_buffer);

// Function to initialize 135-bit value from a hexadecimal string
__host__ void initialize_135_bit_value_from_hex(const char *hex_str, unsigned char &high, unsigned long long &mid, unsigned long long &low)
{
    // hex_str should be 34 hex digits (for 135 bits)
    if (strlen(hex_str) < 34)
    {
        fprintf(stderr, "Hex string too short: %s\n", hex_str);
        exit(1);
    }

    // Parse the 'high' part (first 2 hex digits, but only 7 bits are used)
    char high_str[3] = { hex_str[0], hex_str[1], '\0' };
    high = (unsigned char)(strtoul(high_str, NULL, 16) & 0x7F); // Only 7 bits

    // Parse the 'mid' part (next 16 hex digits)
    char mid_str[17];
    strncpy(mid_str, &hex_str[2], 16);
    mid_str[16] = '\0';
    mid = strtoull(mid_str, NULL, 16);

    // Parse the 'low' part (last 16 hex digits)
    char low_str[17];
    strncpy(low_str, &hex_str[18], 16);
    low_str[16] = '\0';
    low = strtoull(low_str, NULL, 16);
    
    // Since the hex string represents the number in big-endian order,
    // convert 'mid' and 'low' to little-endian format to match storage.
    // Swap bytes of 'mid' and 'low' to convert to little-endian.
    unsigned long long mid_le = 0;
    unsigned long long low_le = 0;
    for (int i = 0; i < 8; i++) {
        mid_le |= ((mid >> (i * 8)) & 0xFF) << ((7 - i) * 8);
        low_le |= ((low >> (i * 8)) & 0xFF) << ((7 - i) * 8);
    }
    mid = mid_le;
    low = low_le;
}

// Corrected FNV-1a hash function for points (little-endian order)
__host__ __device__ __forceinline__ unsigned long long fnv1a_hash_point(const unsigned char high, const unsigned long long mid, const unsigned long long low)
{
    const unsigned long long FNV_prime = 1099511628211ULL;
    unsigned long long hash = 14695981039346656037ULL; // FNV offset basis

    // Hash the 'high' byte
    hash ^= high;
    hash *= FNV_prime;

    // Hash the 'mid' 8 bytes in little-endian order
    for (int i = 0; i < 8; i++)
    {
        hash ^= (mid >> (i * 8)) & 0xFF;
        hash *= FNV_prime;
    }

    // Hash the 'low' 8 bytes in little-endian order
    for (int i = 0; i < 8; i++)
    {
        hash ^= (low >> (i * 8)) & 0xFF;
        hash *= FNV_prime;
    }

    return hash;
}

// Function to add a value to Steps128 (Host and Device)
__host__ __device__ void add_to_steps(Steps128 &steps, unsigned long long value)
{
    // Perform 128-bit addition: steps += value
    unsigned long long old_low = steps.low;
    steps.low += value;
    unsigned long long carry = (steps.low < old_low) ? 1ULL : 0ULL;
    steps.high += carry;
}

// Function to add step size to tame value with overflow handling (Host and Device)
__host__ __device__ void add_step_to_tame_value(
    unsigned char &tame_high,        // 7 bits
    unsigned long long &mid,         // 64 bits
    unsigned long long &low,         // 64 bits
    unsigned long long step_size     // 64 bits
)
{
    // Perform 128-bit addition: (mid:low) + step_size
    unsigned long long new_low = low + step_size;
    unsigned long long carry = (new_low < low) ? 1ULL : 0ULL;
    low = new_low;

    unsigned long long new_mid = mid + carry;
    carry = (new_mid < mid) ? 1ULL : 0ULL;
    mid = new_mid;

    // Update high with any carry from mid
    if (carry)
    {
        tame_high = (tame_high + 1) & 0x7F; // Ensure only 7 bits are used
    }
}

// Function to compute step size on the host
unsigned long long compute_step_size_host(const unsigned char high, const unsigned long long mid, const unsigned long long low, const std::vector<unsigned long long>& step_sizes)
{
    unsigned long long hash = fnv1a_hash_point(high, mid, low);
    unsigned int index = hash % NUM_STEP_FUNCTIONS;
    return step_sizes[index];
}

// Device and Host function to check if a Point has 25 trailing zero bits
__host__ __device__ bool has_trailing_zeros(const Point &p) {
    return (p.low & ((1ULL << TARGET_TAIL_BITS) - 1)) == 0;
}

// Define is_distinguished_point using has_trailing_zeros
__host__ __device__ bool is_distinguished_point(unsigned char high, unsigned long long mid, unsigned long long low) {
    Point p;
    p.high = high;
    p.mid = mid;
    p.low = low;
    return has_trailing_zeros(p);
}

// Function to write a tame DP to file
void write_tame_dp_to_file(const Point &p, int device_id, int &file_part_number, unsigned long long &current_file_size, std::vector<Point> &tame_buffer)
{
    // Prepare filename
    char tame_filename[256];
    snprintf(tame_filename, sizeof(tame_filename), "tame_device_%d_part_%d.bin", device_id, file_part_number);

    // Check if writing the buffer would exceed the max file size
    unsigned long long data_size = sizeof(Point);
    if (current_file_size + data_size > FILE_SIZE_CAP) {
        // Start a new tame file
        file_part_number++;
        snprintf(tame_filename, sizeof(tame_filename), "tame_device_%d_part_%d.bin", device_id, file_part_number);
        current_file_size = 0;
    }

    // Open the tame file in append mode
    std::ofstream tame_file(tame_filename, std::ios::binary | std::ios::app);
    if (!tame_file.is_open()) {
        fprintf(stderr, "Failed to open tame file: %s\n", tame_filename);
        return;
    }

    // Write the tame point directly as binary
    if (has_trailing_zeros(p)) { // Ensuring compliance
        tame_file.write(reinterpret_cast<const char*>(&p), sizeof(Point));
    } else {
        // Optionally, handle non-compliant Points if needed
    }
    if (tame_file) {
        current_file_size += data_size;
    } else {
        fprintf(stderr, "Failed to write to tame file: %s\n", tame_filename);
    }

    tame_file.close();
}

// Function to compute the tame path on the host
void compute_tame_path(
    const char* initial_value,
    std::unordered_set<Point, PointHash, PointEqual> &tame_points,
    std::atomic<bool> &collision_found,
    std::mutex &dp_mutex,
    Point &last_tame_dp
)
{
    // Initialize the 135-bit tame value
    unsigned char tame_high;
    unsigned long long mid, low;
    initialize_135_bit_value_from_hex(initial_value, tame_high, mid, low);

    Steps128 steps_tame = {0ULL, 0ULL};

    // Prepare step sizes array
    std::vector<unsigned long long> step_sizes(NUM_STEP_FUNCTIONS);
    unsigned int jumpBit = 10; // Adjusted jumpBit to 10 for smaller step sizes

    // Calculate minAvg and maxAvg
    double minAvg = pow(2.0, (double)jumpBit - 1.05);
    double maxAvg = pow(2.0, (double)jumpBit - 0.95);

    // Generate step sizes between minAvg and maxAvg
    for (int i = 0; i < NUM_STEP_FUNCTIONS; ++i) {
        double fraction = (double)i / (NUM_STEP_FUNCTIONS - 1);
        double step_size = minAvg + fraction * (maxAvg - minAvg);
        unsigned long long step = (unsigned long long)step_size;

        step_sizes[i] = step;
    }

    bool is_first_point = true; // Flag to skip initial DP detection if undesired

    // Variables for writing tame DPs
    int device_id = 0; // Assuming device_id=0 for tame DPs
    int file_part_number = 0;
    unsigned long long tame_current_file_size = 0;
    std::vector<Point> tame_buffer;
    tame_buffer.reserve(BUFFER_SIZE);

    // Iterate until collision is found
    while (!collision_found.load())
    {
        // Compute step size based on the current point
        unsigned long long step_size = compute_step_size_host(tame_high, mid, low, step_sizes);

        // Update the tame point by adding step_size
        add_step_to_tame_value(tame_high, mid, low, step_size);
        add_to_steps(steps_tame, step_size);

        // Update the last tame DP
        {
            std::lock_guard<std::mutex> lock(dp_mutex);
            last_tame_dp.high = tame_high;
            last_tame_dp.mid = mid;
            last_tame_dp.low = low;
            last_tame_dp.steps = steps_tame;
            last_tame_dp.is_tame = 1;
        }

        // Check if tame point is distinguished
        if (is_distinguished_point(tame_high, mid, low))
        {
            if (is_first_point) {
                is_first_point = false;
                // Optionally, skip the first point if undesired
            }

            Point p;
            p.high = tame_high;
            p.mid = mid;
            p.low = low;
            p.steps = steps_tame;
            p.is_tame = 1;

            // Add to tame_points set
            {
                std::lock_guard<std::mutex> lock(dp_mutex);
                tame_points.insert(p);
                last_tame_dp = p;
            }

            // Write the tame DP to binary file
            write_tame_dp_to_file(p, device_id, file_part_number, tame_current_file_size, tame_buffer);
        }
    }

    // Flush any remaining tame DPs to disk
    if (!tame_buffer.empty()) {
        write_tame_dp_to_file(tame_buffer.back(), device_id, file_part_number, tame_current_file_size, tame_buffer);
    }
}

// Collision detection thread function
void collision_detection_thread(
    ThreadSafeQueue<Point> &collision_queue,
    std::unordered_set<Point, PointHash, PointEqual> &tame_points,
    std::mutex &collision_mutex,
    std::atomic<bool> &collision_found
)
{
    Point point;

    while (!collision_found.load()) {
        if (collision_queue.dequeue(point)) {
            std::lock_guard<std::mutex> lock(collision_mutex);

            // Check if the wild point matches any tame point
            auto it = tame_points.find(point);
            if (it != tame_points.end()) {
                // Collision found
                collision_found.store(true);
                std::cout << "Collision detected!" << std::endl;
                std::cout << "Wild Point:" << std::endl;
                std::cout << "  high: 0x" << std::hex << static_cast<int>(point.high) << std::dec << std::endl;
                std::cout << "  mid:  0x" << std::hex << point.mid << std::dec << std::endl;
                std::cout << "  low:  0x" << std::hex << point.low << std::dec << std::endl;
                break;
            }
        }
    }
}

// Kernel to initialize curand states
__global__ void init_curand_states_kernel(hiprandState *state, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize each state with a unique seed based on thread index and device seed
    hiprand_init(seed + idx, 0, 0, &state[idx]);
}

// Kernel to generate wild paths and collect distinguished points
__global__ void generate_wild_paths_kernel(
    hiprandState *state,
    Counter128 *global_counter,
    Point *dp_points,
    unsigned int *dp_count,
    unsigned long long batch_size,
    unsigned int max_dp,
    unsigned long long* d_step_sizes
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = state[idx];

    // Initialize local wild variables
    unsigned char local_wild_high;
    unsigned long long local_wild_mid, local_wild_low;

    // Initialize wild point with high nibble '4' and random low nibble (0-F)
    local_wild_high = 0x40 | (hiprand(&localState) & 0x0F);

    // Generate random 'mid' and 'low' values
    local_wild_mid = ((unsigned long long)hiprand(&localState) << 32) | hiprand(&localState);
    local_wild_low = ((unsigned long long)hiprand(&localState) << 32) | hiprand(&localState);

    Steps128 steps_wild = {0ULL, 0ULL};

    for (unsigned long long batch = 0; batch < batch_size; ++batch)
    {
        // Compute step size based on the current point
        unsigned long long hash = fnv1a_hash_point(local_wild_high, local_wild_mid, local_wild_low);
        unsigned long long index = hash % NUM_STEP_FUNCTIONS;
        unsigned long long step_size = d_step_sizes[index];

        // Update the wild point
        unsigned long long new_low = local_wild_low + step_size;
        unsigned long long carry = (new_low < local_wild_low) ? 1ULL : 0ULL;
        local_wild_low = new_low;

        unsigned long long new_mid = local_wild_mid + carry;
        carry = (new_mid < local_wild_mid) ? 1ULL : 0ULL;
        local_wild_mid = new_mid;

        if (carry)
        {
            local_wild_high = (local_wild_high + 1) & 0x7F; // Ensure only 7 bits are used
        }

        add_to_steps(steps_wild, step_size);

        // Check if wild point is distinguished
        if (is_distinguished_point(local_wild_high, local_wild_mid, local_wild_low))
        {
            // Atomically fetch and add
            unsigned int dp_idx = atomicAdd(dp_count, 1);
            if (dp_idx < max_dp) {
                dp_points[dp_idx].high = local_wild_high;
                dp_points[dp_idx].mid = local_wild_mid;
                dp_points[dp_idx].low = local_wild_low;
                dp_points[dp_idx].steps = steps_wild;
                dp_points[dp_idx].is_tame = 0;
            }
            else {
                // Reached max_dp, exit early
                break;
            }
        }

        if (*dp_count >= max_dp) {
            break;
        }
    }

    // Update the global 128-bit counter
    atomicAdd128(global_counter, steps_wild);

    // Save the updated state
    state[idx] = localState;
}

// Function to get file size
unsigned long long get_file_size(const char* filename) {
    struct stat st;
    if (stat(filename, &st) != 0) {
        // File does not exist yet
        return 0;
    }
    return static_cast<unsigned long long>(st.st_size);
}

// Step monitor function
void step_monitor(Counter128 *global_counter, std::vector<StepThreshold> &thresholds, std::atomic<bool> &stop_flag, std::mutex &dp_mutex, Point &last_tame_dp, Point &last_wild_dp, std::atomic<unsigned long long> &kernel_launch_count, int device_count)
{
    bool all_printed = false;
    auto last_time = std::chrono::steady_clock::now();
    unsigned long long last_steps = 0;

    while (!all_printed && !stop_flag.load())
    {
        // Read the current total steps atomically
        unsigned long long high = global_counter->high;
        unsigned long long low = global_counter->low;

        // Calculate the current step count as a 128-bit number
        unsigned __int128 total_steps = ((unsigned __int128)high << 64) | low;

        // Calculate expected steps
        unsigned long long launches = kernel_launch_count.load();
        unsigned long long expected_steps = launches * BATCH_SIZE * THREADS_PER_BLOCK * BLOCKS_PER_GPU * device_count;

        // Compare total_steps with expected_steps
        if (total_steps < expected_steps) {
            printf("Warning: Total steps (%llu) less than expected steps (%llu).\n", (unsigned long long)total_steps, expected_steps);
        } else if (total_steps > expected_steps) {
            printf("Warning: Total steps (%llu) exceed expected steps (%llu).\n", (unsigned long long)total_steps, expected_steps);
        } else {
            printf("Step counter is accurate. Total steps: %llu\n", (unsigned long long)total_steps);
        }

        // Iterate through thresholds
        for (auto &threshold : thresholds)
        {
            unsigned __int128 threshold_value;
            if (threshold.exponent < 64) {
                threshold_value = ((unsigned __int128)0 << 64) | ((1ULL) << threshold.exponent);
            }
            else {
                if (threshold.exponent - 64 < 64) {
                    threshold_value = ((unsigned __int128)(1ULL << (threshold.exponent - 64)) << 64) | 0;
                }
                else {
                    threshold_value = 0; // Exceeds 128-bit representation
                }
            }

            if (!threshold.printed && total_steps >= threshold_value)
            {
                // Print the threshold reached
                printf("Total steps reached 2^%u.\n", threshold.exponent);
                threshold.printed = true;
            }
        }

        // Calculate steps per second
        auto current_time = std::chrono::steady_clock::now();
        std::chrono::duration<double> elapsed_seconds = current_time - last_time;
        unsigned long long current_steps = (unsigned long long)total_steps;
        unsigned long long steps_diff = current_steps - last_steps;
        double steps_per_second = elapsed_seconds.count() > 0 ? steps_diff / elapsed_seconds.count() : 0.0;

        printf("Current total steps: %llu (low), %llu (high)\n", low, high);
        printf("Steps in the last %.2f seconds: %llu (%.2f steps/sec)\n\n", elapsed_seconds.count(), steps_diff, steps_per_second);

        // Update for next iteration
        last_time = current_time;
        last_steps = current_steps;

        // Check if all thresholds have been printed
        all_printed = true;
        for (const auto &threshold : thresholds)
        {
            if (!threshold.printed)
            {
                all_printed = false;
                break;
            }
        }

        // Sleep for ten seconds
        sleep(10);
    }
}

// Function to run wild paths on each GPU
void run_on_device(
    int device_id,
    Counter128 *global_counter,
    std::atomic<unsigned long long> &global_dp_counter,
    ThreadSafeQueue<Point> &collision_queue,
    std::atomic<bool> &collision_found,
    std::mutex &dp_mutex,
    Point &last_wild_dp,
    std::atomic<unsigned long long> &kernel_launch_count
)
{
    hipError_t err;

    // Set the current device
    err = hipSetDevice(device_id);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to set device: %s\n", device_id, hipGetErrorString(err));
        return;
    }

    // Allocate memory for curand states
    hiprandState *d_state;
    err = hipMalloc(&d_state, THREADS_PER_BLOCK * BLOCKS_PER_GPU * sizeof(hiprandState));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to allocate device state: %s\n", device_id, hipGetErrorString(err));
        return;
    }

    // Allocate device memory for distinguished points
    Point *d_dp_points;
    err = hipMalloc(&d_dp_points, MAX_DISTINGUISHED_POINTS_PER_KERNEL * sizeof(Point));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to allocate distinguished points memory: %s\n", device_id, hipGetErrorString(err));
        hipFree(d_state);
        return;
    }

    // Allocate device memory for dp_count
    unsigned int *d_dp_count;
    err = hipMalloc(&d_dp_count, sizeof(unsigned int));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to allocate distinguished points count memory: %s\n", device_id, hipGetErrorString(err));
        hipFree(d_state);
        hipFree(d_dp_points);
        return;
    }
    err = hipMemset(d_dp_count, 0, sizeof(unsigned int));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to reset dp_count: %s\n", device_id, hipGetErrorString(err));
        hipFree(d_state);
        hipFree(d_dp_points);
        hipFree(d_dp_count);
        return;
    }

    // Allocate pinned host memory for dps
    Point *h_dp_points;
    err = hipHostAlloc(&h_dp_points, MAX_DISTINGUISHED_POINTS_PER_KERNEL * sizeof(Point), hipHostMallocDefault);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to allocate pinned host memory: %s\n", device_id, hipGetErrorString(err));
        hipFree(d_state);
        hipFree(d_dp_points);
        hipFree(d_dp_count);
        return;
    }

    // Initialize curand states with unique seed per GPU and thread
    unsigned long long base_seed = SEED + (device_id * 1000000);

    // Initialize curand states
    init_curand_states_kernel<<<BLOCKS_PER_GPU, THREADS_PER_BLOCK>>>(d_state, base_seed);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to launch init_curand_states kernel: %s\n", device_id, hipGetErrorString(err));
        hipFree(d_state);
        hipFree(d_dp_points);
        hipFree(d_dp_count);
        hipHostFree(h_dp_points);
        return;
    }

    // Synchronize to ensure curand states are initialized
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: CUDA Device Synchronize failed after init_curand_states: %s\n", device_id, hipGetErrorString(err));
        hipFree(d_state);
        hipFree(d_dp_points);
        hipFree(d_dp_count);
        hipHostFree(h_dp_points);
        return;
    }

    // Prepare step sizes array
    unsigned long long h_step_sizes[NUM_STEP_FUNCTIONS];
    unsigned int jumpBit = 20; // Adjusted jumpBit to 20 for smaller step sizes

    // Calculate minAvg and maxAvg
    double minAvg = pow(2.0, (double)jumpBit - 1.05);
    double maxAvg = pow(2.0, (double)jumpBit - 0.95);

    // Generate step sizes between minAvg and maxAvg
    for (int i = 0; i < NUM_STEP_FUNCTIONS; ++i) {
        double fraction = (double)i / (NUM_STEP_FUNCTIONS - 1);
        double step_size = minAvg + fraction * (maxAvg - minAvg);
        unsigned long long step = (unsigned long long)step_size;

        h_step_sizes[i] = step;
    }

    // Copy step sizes to device
    unsigned long long* d_step_sizes;
    err = hipMalloc(&d_step_sizes, NUM_STEP_FUNCTIONS * sizeof(unsigned long long));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to allocate device step sizes: %s\n", device_id, hipGetErrorString(err));
        hipFree(d_state);
        hipFree(d_dp_points);
        hipFree(d_dp_count);
        hipHostFree(h_dp_points);
        return;
    }
    err = hipMemcpy(d_step_sizes, h_step_sizes, NUM_STEP_FUNCTIONS * sizeof(unsigned long long), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU %d: Failed to copy step sizes to device: %s\n", device_id, hipGetErrorString(err));
        hipFree(d_state);
        hipFree(d_dp_points);
        hipFree(d_dp_count);
        hipHostFree(h_dp_points);
        hipFree(d_step_sizes);
        return;
    }

    // Launch parameters
    unsigned long long iterations_per_launch = BATCH_SIZE;
    dim3 grid(BLOCKS_PER_GPU);
    dim3 block(THREADS_PER_BLOCK);

    // Variables to keep track of file size and part number
    unsigned long long current_file_size = 0;
    int file_part_number = 0;

    // Prepare initial filename for wild DPs
    char wild_filename[256];
    snprintf(wild_filename, sizeof(wild_filename), "wild_device_%d_part_%d.bin", device_id, file_part_number);

    // Get the current file size
    current_file_size = get_file_size(wild_filename);

    // Initialize buffer to store wild DPs before writing
    std::vector<Point> wild_buffer;
    wild_buffer.reserve(BUFFER_SIZE);

    // Main loop controlled by the host
    while (!collision_found.load())
    {
        // Reset dp_count on device
        err = hipMemset(d_dp_count, 0, sizeof(unsigned int));
        if (err != hipSuccess) {
            fprintf(stderr, "GPU %d: Failed to reset dp_count: %s\n", device_id, hipGetErrorString(err));
            break;
        }

        // Launch the generate_wild_paths kernel with max_dp parameter
        generate_wild_paths_kernel<<<grid, block>>>(
            d_state,
            global_counter,
            d_dp_points,
            d_dp_count,
            iterations_per_launch,
            MAX_DISTINGUISHED_POINTS_PER_KERNEL,
            d_step_sizes
        );
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "GPU %d: Failed to launch generate_wild_paths kernel: %s\n", device_id, hipGetErrorString(err));
            break;
        }

        // Synchronize to wait for kernel completion
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "GPU %d: CUDA Device Synchronize failed after generate_wild_paths: %s\n", device_id, hipGetErrorString(err));
            break;
        }

        // Increment kernel launch count
        kernel_launch_count.fetch_add(1);

        // Copy dp_count from device to host
        unsigned int dp_count_host;
        err = hipMemcpy(&dp_count_host, d_dp_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr, "GPU %d: Failed to copy dp_count from device to host: %s\n", device_id, hipGetErrorString(err));
            break;
        }

        // Clamp dp_count_host to max_dp
        if (dp_count_host > MAX_DISTINGUISHED_POINTS_PER_KERNEL) {
            dp_count_host = MAX_DISTINGUISHED_POINTS_PER_KERNEL;
        }

        if (dp_count_host > 0) {
            // Copy DPs from device to host pinned memory
            err = hipMemcpy(h_dp_points, d_dp_points, dp_count_host * sizeof(Point), hipMemcpyDeviceToHost);
            if (err != hipSuccess) {
                fprintf(stderr, "GPU %d: Failed to copy distinguished points from device to host: %s\n", device_id, hipGetErrorString(err));
                break;
            }

            // Buffer the wild DPs
            for (unsigned int i = 0; i < dp_count_host; ++i) {
                // Increment the global DP counter atomically
                unsigned long long current_count = global_dp_counter.fetch_add(1) + 1;

                // Add the DP to the wild buffer
                wild_buffer.push_back(h_dp_points[i]);

                // Update the last wild DP
                {
                    std::lock_guard<std::mutex> lock(dp_mutex);
                    last_wild_dp = h_dp_points[i];
                }

                // Enqueue the wild DP for collision detection
                collision_queue.enqueue(h_dp_points[i]);

                // If wild buffer is full, write it to disk as binary
                if (wild_buffer.size() >= BUFFER_SIZE) {
                    // Check if writing the buffer would exceed the max file size
                    unsigned long long data_size = wild_buffer.size() * sizeof(Point);
                    if (current_file_size + data_size > FILE_SIZE_CAP) {
                        // Start a new wild file
                        file_part_number++;
                        snprintf(wild_filename, sizeof(wild_filename), "wild_device_%d_part_%d.bin", device_id, file_part_number);
                        current_file_size = 0;
                    }

                    // Open the wild file in append mode
                    std::ofstream wild_file(wild_filename, std::ios::binary | std::ios::app);
                    if (!wild_file.is_open()) {
                        fprintf(stderr, "GPU %d: Failed to open wild file: %s\n", device_id, wild_filename);
                        wild_buffer.clear();
                        wild_buffer.reserve(BUFFER_SIZE);
                        continue;
                    }

                    // Write the buffered wild points directly as binary
                    for (const auto& p : wild_buffer) {
                        if (has_trailing_zeros(p)) {
                            wild_file.write(reinterpret_cast<const char*>(&p), sizeof(Point));
                        } else {
                            // Optionally, handle non-compliant Points if needed
                        }
                    }
                    if (wild_file) {
                        current_file_size += data_size;
                    } else {
                        fprintf(stderr, "GPU %d: Failed to write to wild file: %s\n", device_id, wild_filename);
                    }

                    wild_file.close();
                    wild_buffer.clear();
                    wild_buffer.reserve(BUFFER_SIZE);
                }

                // Check if a collision has been found to terminate early
                if (collision_found.load()) {
                    break;
                }
            }

            // If a collision has been found, terminate the loop
            if (collision_found.load()) {
                break;
            }
        }
    }

    // Cleanup resources
    hipFree(d_state);
    hipFree(d_dp_points);
    hipFree(d_dp_count);
    hipHostFree(h_dp_points);
    hipFree(d_step_sizes);
}

// Main Function
int main()
{
    int device_count;
    hipError_t err = hipGetDeviceCount(&device_count);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device count: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (device_count < 1) {
        fprintf(stderr, "No CUDA devices found.\n");
        return 1;
    }

    printf("Found %d CUDA device(s).\n", device_count);

    // Allocate unified memory for the global counter
    Counter128 *global_counter;
    err = hipMallocManaged(&global_counter, sizeof(Counter128));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate global counter memory: %s\n", hipGetErrorString(err));
        return 1;
    }
    global_counter->low = 0;
    global_counter->high = 0;

    // Define the step thresholds dynamically from 2^60 to 2^127
    std::vector<StepThreshold> thresholds;

    // Starting exponent
    unsigned int start_exponent = 60;
    // Maximum exponent (up to 127 for 128-bit counter)
    unsigned int max_exponent = 127;

    for (unsigned int exponent = start_exponent; exponent <= max_exponent; ++exponent)
    {
        StepThreshold threshold;
        threshold.exponent = exponent;
        threshold.printed = false;

        if (exponent < 64)
        {
            threshold.high = 0;
            threshold.low = 1ULL << exponent;
        }
        else
        {
            // For exponents >= 64, set the high part accordingly
            // Handle exponents up to 127 to prevent overflow
            if (exponent - 64 < 64)
            {
                threshold.high = 1ULL << (exponent - 64);
                threshold.low = 0;
            }
            else
            {
                threshold.high = 0;
                threshold.low = 0;
                fprintf(stderr, "Exponent %u is too large to represent in 128 bits.\n", exponent);
                continue;
            }
        }

        thresholds.push_back(threshold);
    }

    // Create a thread for step monitoring
    std::atomic<bool> stop_monitor(false);

    // Initialize last tame and wild DPs
    Point last_tame_dp = {};
    Point last_wild_dp = {};

    // Mutex to protect last DP variables
    std::mutex dp_mutex;

    // Initialize kernel launch count
    std::atomic<unsigned long long> kernel_launch_count(0);

    // Create a thread-safe queue for collision detection
    ThreadSafeQueue<Point> collision_queue;

    // Create global counters for DPs
    std::atomic<unsigned long long> global_dp_counter(0);

    // Create a mutex for collision detection
    std::mutex collision_mutex;

    // Create an unordered_set for tame points
    std::unordered_set<Point, PointHash, PointEqual> tame_points;

    // Create an atomic flag to indicate if a collision was found
    std::atomic<bool> collision_found(false);

    // Start the step monitor thread
    std::thread monitor_thread(
        step_monitor, 
        global_counter, 
        std::ref(thresholds), 
        std::ref(stop_monitor), 
        std::ref(dp_mutex), 
        std::ref(last_tame_dp), 
        std::ref(last_wild_dp), 
        std::ref(kernel_launch_count), 
        device_count
    );

    // Create threads for each GPU to run wild paths
    std::vector<std::thread> device_threads;
    for (int device_id = 0; device_id < device_count; ++device_id)
    {
        device_threads.emplace_back(
            run_on_device, 
            device_id, 
            global_counter, 
            std::ref(global_dp_counter), 
            std::ref(collision_queue), 
            std::ref(collision_found), 
            std::ref(dp_mutex), 
            std::ref(last_wild_dp), 
            std::ref(kernel_launch_count)
        );
    }

    // Start the tame path computation on a separate thread
    std::thread tame_thread(
        compute_tame_path, 
        INITIAL_VALUE, 
        std::ref(tame_points), 
        std::ref(collision_found), 
        std::ref(dp_mutex), 
        std::ref(last_tame_dp)
    );

    // Create a collision detection thread
    std::thread collision_thread(
        collision_detection_thread, 
        std::ref(collision_queue), 
        std::ref(tame_points), 
        std::ref(collision_mutex), 
        std::ref(collision_found)
    );

    // Wait for the collision detection thread to finish (i.e., a collision is found)
    collision_thread.join();

    // Signal device threads and tame thread to stop by setting collision_found to true
    collision_found.store(true);

    // Wait for all device threads to finish
    for (auto &t : device_threads)
    {
        if (t.joinable()) {
            t.join();
        }
    }

    // Wait for the tame thread to finish
    if (tame_thread.joinable()) {
        tame_thread.join();
    }

    // Stop the monitor thread
    stop_monitor.store(true);
    if (monitor_thread.joinable()) {
        monitor_thread.join();
    }

    // Free the global counter memory
    hipFree(global_counter);

    printf("Program terminated after detecting a collision.\n");

    return 0;
}
